
#include <hip/hip_runtime.h>
__global__ void naive_matmul(char* aPtr, char* bPtr, char* cPtr, int m, int n, int k) \
{
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int x = threadIdx.x + blockIdx.x*blockDim.x;

    if (y >= m || x >= n) return;


    // precomputing values
    char* aValOffset = aPtr + y*k*4;
    char* bValOffset = bPtr + x*4;
    float cVal = 0;

    for (int i = 0; i < k; ++i) {
    
        float aVal = *(float*)aValOffset;
        float bVal = *(float*)bValOffset;

        cVal += aVal * bVal;

        aValOffset += 4;
        bValOffset += 4*n;
    }

    // set c value
    // compute linear offset
    char* outPtr = cPtr + (y*n + x)*4;
    *(float*)outPtr = cVal;
}